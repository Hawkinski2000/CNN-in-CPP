#include "hip/hip_runtime.h"
#include <iostream>
#include "Tensor.h"


__global__ void maxpool2d_kernel(const float* input,
                                 float* output,
                                 int N,
                                 int C,
                                 int in_H,
                                 int in_W,
                                 int out_H,
                                 int out_W,
                                 int kH,
                                 int kW,
                                 int stride,
                                 int padding,
                                 int dilation) {
    int n = blockIdx.x; // Batch index
    int c = blockIdx.y; // Input channel index

    int out_h = threadIdx.y; // Output height position
    int out_w = threadIdx.x; // Output width position

    if (out_h >= out_H || out_w >= out_W) {
        return;
    }

    // The top-left corner of this patch
    int in_i = out_h * stride - padding;
    int in_j = out_w * stride - padding;

    float max_val = -INFINITY;

    for (int kh = 0; kh < kH; kh++) {
        for (int kw = 0; kw < kW; kw++) {
            int in_h = in_i + kh * dilation; // Input height position
            int in_w = in_j + kw * dilation; // Input width position

            if (in_h >= 0 && in_h < in_H && in_w >= 0 && in_w < in_W) {
                int input_idx = ((n * C + c) * in_H + in_h) * in_W + in_w;
                max_val = max(max_val, input[input_idx]);
            }
        }
    }

    int output_idx = ((n * C + c) * out_H + out_h) * out_W + out_w;
    output[output_idx] = max_val;
}

Tensor maxpool2d_cuda(Tensor input, initializer_list<size_t> kernel_size, size_t stride, size_t padding, size_t dilation) {
    size_t kH = *kernel_size.begin(); // Kernel height
    size_t kW; // Kernel width
    if (kernel_size.size() == 1) {
        kW = kH;
    }
    else {
        kW = *(kernel_size.begin() + 1);
    }

    if (stride == 0) {
        stride = kH;
    }

    size_t N = input.dimensions[0]; // Batch size

    size_t C = input.dimensions[1]; // Channels

    size_t in_H = input.dimensions[2]; // Input height
    size_t in_W = input.dimensions[3]; // Input width

    size_t out_H = ((in_H + 2 * padding - dilation * (kH - 1) - 1) / stride) + 1; // Output height
    size_t out_W = ((in_W + 2 * padding - dilation * (kW - 1) - 1) / stride) + 1; // Output width

    Tensor result = Tensor::empty({N, C, out_H, out_W}, true);

    dim3 gridDim(N, C);
    dim3 blockDim(out_W, out_H);

    // Allocate GPU memory for the input tensor if not already
    if (!input.device_data) {
        hipMalloc(&input.device_data, input.total_elements * sizeof(float));
    }

    // Transfer the input tensor's data from CPU to GPU
    hipMemcpy(input.device_data, input.data.get(), sizeof(float) * input.total_elements, hipMemcpyHostToDevice);

    maxpool2d_kernel<<<gridDim, blockDim>>>(input.device_data,
                                            result.device_data,
                                            N,
                                            C,
                                            in_H,
                                            in_W,
                                            out_H,
                                            out_W,
                                            kH,
                                            kW,
                                            stride,
                                            padding,
                                            dilation);

    hipDeviceSynchronize();

    // Transfer the result tensor's data from GPU to CPU
    hipMemcpy(result.data.get(), result.device_data, sizeof(float) * result.total_elements, hipMemcpyDeviceToHost);

    return result;
}
