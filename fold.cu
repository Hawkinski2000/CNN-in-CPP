#include "hip/hip_runtime.h"
#include <iostream>
#include "Tensor.h"


__global__ void fold_kernel(const float* input,
                            float* output,
                            int N,
                            int C,
                            int out_H,
                            int out_W,
                            int kH,
                            int kW,
                            int stride,
                            int padding,
                            int dilation,
                            int L) {
    int n = blockIdx.x; // Batch index
    int l = blockIdx.y; // Output column
    int patch_idx = threadIdx.x; // Index within each patch vector

    if (patch_idx >= C * kH * kW) {
        return;
    }

    // The top-left corner of this patch
    int out_i = l / out_W;
    int out_j = l % out_W;

    int c = patch_idx / (kH * kW); // Input channel index
    int kh = (patch_idx / kW) % kH; // Kernel height position
    int kw = patch_idx % kW; // Kernel width position

    int out_h = out_i * stride - padding + kh * dilation; // output height position
    int out_w = out_j * stride - padding + kw * dilation; // output width position

    if (out_h < 0 || out_h >= out_H || out_w < 0 || out_w >= out_W) {
        return;
    }

    int input_idx = ((n * (C * kH * kW) + patch_idx) * L) + l; // Index in the input tensor
    int output_idx = ((n * C + c) * out_H + out_h) * out_W + out_w; // Index in the folded output tensor

    atomicAdd(&output[output_idx], input[input_idx]);
}

Tensor fold_cuda(Tensor& input, initializer_list<size_t> output_size, initializer_list<size_t> kernel_size, size_t dilation, size_t padding, size_t stride) {
    size_t kH = *kernel_size.begin(); // Kernel height
    size_t kW; // Kernel width
    if (kernel_size.size() == 1) {
        kW = kH;
    }
    else {
        kW = *(kernel_size.begin() + 1);
    }

    size_t out_H = *output_size.begin(); // Output height
    size_t out_W = *(output_size.begin() + 1); // Output width

    size_t N = input.dimensions[0]; // Batch size

    size_t C = input.dimensions[1] / (kH * kW); // Channels

    size_t L = input.dimensions[2]; // Total number of blocks

    Tensor result = Tensor::zeros({N, C, out_H, out_W}, true);

    dim3 gridDim(static_cast<unsigned int>(N), static_cast<unsigned int>(L));
    dim3 blockDim(static_cast<unsigned int>(C * kH * kW));

    // Allocate GPU memory for the input tensor if not already
    if (!input.device_data) {
        hipMalloc(&input.device_data, input.total_elements * sizeof(float));
    }

    // Transfer the input tensor's data from CPU to GPU
    hipMemcpy(input.device_data, input.data.get(), sizeof(float) * input.total_elements, hipMemcpyHostToDevice);

    fold_kernel<<<gridDim, blockDim>>>(input.device_data,
                                       result.device_data,
                                       N,
                                       C,
                                       out_H,
                                       out_W,
                                       kH,
                                       kW,
                                       stride,
                                       padding,
                                       dilation,
                                       L);

    hipDeviceSynchronize();

    // Transfer the result tensor's data from GPU to CPU
    hipMemcpy(result.data.get(), result.device_data, sizeof(float) * result.total_elements, hipMemcpyDeviceToHost);

    return result;
}
