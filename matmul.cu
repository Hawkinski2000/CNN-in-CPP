#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "Tensor.h"


/*
==============================================================================
This function uses code from Simon Boehm's repository, "SGEMM_CUDA":
    https://github.com/siboehm/SGEMM_CUDA/tree/master
==============================================================================
*/

Tensor Tensor::matmul(const Tensor& other) {
    hipSetDevice(0);
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    size_t m, n, k;
    m = dimensions[dimensions.size() - 2];
    k = dimensions[dimensions.size() - 1];
    n = other.dimensions[other.dimensions.size() - 1];

    vector<size_t> batch_dims;
    if (dimensions.size() > 2) {
        batch_dims = vector<size_t>(dimensions.begin(), dimensions.end() - 2);
    } else {
        batch_dims = {1};
    }

    size_t batch_count = 1;
    for (size_t dim : batch_dims) {
        batch_count *= dim;
    }

    float alpha = 1, beta = 0; // GEMM input parameters, C=α*AB+β*C

    float *dA, *dB, *dC;

    float* A = data.get();
    float* B = other.data.get();
    float* C = new float[batch_count * m * n];

    hipMalloc((void **)&dA, sizeof(float) * batch_count * m * k);
    hipMalloc((void **)&dB, sizeof(float) * batch_count * k * n);
    hipMalloc((void **)&dC, sizeof(float) * batch_count * m * n);

    hipMemcpy(dA, A, sizeof(float) * batch_count * m * k, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(float) * batch_count * k * n, hipMemcpyHostToDevice);
    hipMemcpy(dC, C, sizeof(float) * batch_count * m * n, hipMemcpyHostToDevice);

    hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, dB, HIP_R_32F,
                n, (k * n), dA, HIP_R_32F, k, (m * k), &beta, dC, HIP_R_32F, n, (m * n), batch_count,
                HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    hipDeviceSynchronize();
    hipMemcpy(C, dC, sizeof(float) * batch_count * m * n, hipMemcpyDeviceToHost);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipblasDestroy(handle);

    Tensor result;
    if (batch_count > 1) {
        result = Tensor::empty({batch_count, m, n});
    }
    else {
        result = Tensor::empty({m, n});
    }
    result.data = shared_ptr<float>(C, default_delete<float[]>());

    return result;
}
