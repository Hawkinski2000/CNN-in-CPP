#include <iostream>
#include <hipblas.h>
#include "Tensor.h"
#include "Node.h"


/*
==============================================================================
This function uses code from Simon Boehm's repository, "SGEMM_CUDA":
    https://github.com/siboehm/SGEMM_CUDA/tree/master
==============================================================================
*/


Tensor Tensor::matmul(Tensor& other, bool transpose_a, bool transpose_b, bool create_node) {
    hipSetDevice(0);

    static hipblasHandle_t handle;
    static bool initialized = false;
    if (!initialized) {
        hipblasCreate(&handle);
        initialized = true;
    }

    hipblasOperation_t transa;
    hipblasOperation_t transb;
    if (transpose_a) {
        transa = HIPBLAS_OP_T;
    }
    else {
        transa = HIPBLAS_OP_N;
    }
    if (transpose_b) {
        transb = HIPBLAS_OP_T;
    }
    else {
        transb = HIPBLAS_OP_N;
    }

    if (dimensions.size() == 1) {
        dimensions = {1, dimensions[0]};
        strides = {dimensions[0], 1};
    }
    if (other.dimensions.size() == 1) {
        other.dimensions = {1, other.dimensions[0]};
        other.strides = {other.dimensions[0], 1};
    }

    size_t A_rows_orig = dimensions[dimensions.size() - 2];
    size_t A_cols_orig = dimensions[dimensions.size() - 1];
    size_t B_rows_orig = other.dimensions[other.dimensions.size() - 2];
    size_t B_cols_orig = other.dimensions[other.dimensions.size() - 1];

    size_t effective_A_rows;
    size_t effective_A_cols;
    size_t effective_B_cols;
    if (transpose_a) {
        effective_A_rows = A_cols_orig;
        effective_A_cols = A_rows_orig;
    }
    else {
        effective_A_rows = A_rows_orig;
        effective_A_cols = A_cols_orig;
    }
    if (transpose_b) {
        effective_B_cols = B_rows_orig;
    }
    else {
        effective_B_cols = B_cols_orig;
    }

    size_t m_result, n_result, k_result;
    m_result = effective_A_rows;
    k_result = effective_A_cols;
    n_result = effective_B_cols;
    size_t m_cublas = n_result;
    size_t n_cublas = m_result;
    size_t k_cublas = k_result;

    size_t lda = B_cols_orig;
    size_t ldb = A_cols_orig;
    size_t ldc = n_result;

    vector<size_t> A_batch_dims;
    if (dimensions.size() > 2) {
        A_batch_dims = vector<size_t>(dimensions.begin(), dimensions.end() - 2);
    } else {
        A_batch_dims = {1};
    }
    vector<size_t> B_batch_dims;
    if (other.dimensions.size() > 2) {
        B_batch_dims = vector<size_t>(other.dimensions.begin(), other.dimensions.end() - 2);
    } else {
        B_batch_dims = {1};
    }

    size_t A_batch_count = 1;
    size_t B_batch_count = 1;
    for (size_t dim : A_batch_dims) {
        A_batch_count *= dim;
    }
    for (size_t dim : B_batch_dims) {
        B_batch_count *= dim;
    }

    size_t strideA = (A_rows_orig * A_cols_orig);
    size_t strideB = (B_rows_orig * B_cols_orig);
    size_t strideC = (m_cublas * n_cublas);
    if (A_batch_count == 1 and B_batch_count > 1) {
        strideA = 0;
    }
    else if (B_batch_count == 1 and A_batch_count > 1) {
        strideB = 0;
    }

    size_t batch_count = 1;
    size_t A_copies = 1;
    size_t B_copies = 1;
    vector<size_t> result_batch_dims;
    if (A_batch_dims != B_batch_dims) {
        if (strideA == 0) {
            batch_count = B_batch_count;
            result_batch_dims = B_batch_dims;
        }
        else if (strideB == 0) {
            batch_count = A_batch_count;
            result_batch_dims = A_batch_dims;
        }
        else {
            result_batch_dims = broadcast_result_shape(A_batch_dims, B_batch_dims);
            for (size_t dim : result_batch_dims) {
                batch_count *= dim;
            }
            A_copies = batch_count / A_batch_count;
            B_copies = batch_count / B_batch_count;
        }
    }
    else {
        batch_count = A_batch_count;
        result_batch_dims = A_batch_dims;
    }

    float *dA, *dB, *dC;

    float* A;
    float* B;
    shared_ptr<float> A_expanded;
    shared_ptr<float> B_expanded;
    if (A_copies > 1) {
        A_expanded = shared_ptr<float>(new float[A_copies * total_elements], default_delete<float[]>());
        for (size_t i = 0; i < A_copies; i++) {
            copy(data.get(), data.get() + total_elements, A_expanded.get() + i * total_elements);
        }
        A = A_expanded.get();
    }
    else {
        A = data.get();
    }
    if (B_copies > 1) {
        B_expanded = shared_ptr<float>(new float[B_copies * other.total_elements], default_delete<float[]>());
        for (size_t i = 0; i < B_copies; i++) {
            copy(other.data.get(), other.data.get() + other.total_elements, B_expanded.get() + i * other.total_elements);
        }
        B = B_expanded.get();
    }
    else {
        B = other.data.get();
    }
    float* C = new float[batch_count * m_result * n_result];

    size_t A_mem_size = sizeof(float) * A_rows_orig * A_cols_orig;
    size_t B_mem_size = sizeof(float) * B_rows_orig * B_cols_orig;
    size_t C_mem_size = sizeof(float) * batch_count * m_cublas * n_cublas;

    if (strideA > 0) {
        A_mem_size *= batch_count;
    }
    if (strideB > 0) {
        B_mem_size *= batch_count;
    }

    hipMalloc((void **)&dA, A_mem_size);
    hipMalloc((void **)&dB, B_mem_size);
    hipMalloc((void **)&dC, C_mem_size);

    hipMemcpy(dA, A, A_mem_size, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, B_mem_size, hipMemcpyHostToDevice);
    hipMemset(dC, 0, C_mem_size);

    float alpha = 1, beta = 0; // GEMM input parameters, C=α*AB+β*C

    hipblasGemmStridedBatchedEx(handle,
                               transb,
                               transa,
                               m_cublas,
                               n_cublas,
                               k_cublas,
                               &alpha,
                               dB,
                               HIP_R_32F,
                               lda,
                               strideB,
                               dA,
                               HIP_R_32F,
                               ldb,
                               strideA,
                               &beta,
                               dC,
                               HIP_R_32F,
                               ldc,
                               strideC,
                               batch_count,
                               HIPBLAS_COMPUTE_32F,
                               CUBLAS_GEMM_DEFAULT_TENSOR_OP);
                             
    hipDeviceSynchronize();
    hipMemcpy(C, dC, C_mem_size, hipMemcpyDeviceToHost);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    Tensor result;
    if (batch_count > 1) {
        vector<size_t> result_dims(result_batch_dims.size() + 2);
        copy(result_batch_dims.begin(), result_batch_dims.end(), result_dims.begin());
        result_dims[result_batch_dims.size()] = m_result;
        result_dims[result_batch_dims.size() + 1] = n_result;
        result = Tensor::empty(result_dims);
    }
    else {
        result = Tensor::empty({m_result, n_result});
    }

    result.data = shared_ptr<float>(C, default_delete<float[]>());
    result.total_elements = batch_count * m_result * n_result;

    if (create_node) {
        if (requires_grad || other.requires_grad) {
            result.node = make_shared<MatmulBackward>(make_shared<Tensor>(*this), make_shared<Tensor>(other));
            result.node->tensor = &result;
        }
    }

    return result;
}
