#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "Tensor.h"


/*
==============================================================================
This function uses code from Simon Boehm's repository, "SGEMM_CUDA":
    https://github.com/siboehm/SGEMM_CUDA/tree/master
==============================================================================
*/

Tensor Tensor::matmul(const Tensor& other) {
    hipSetDevice(0);
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    size_t m, n, k;
    m = dimensions[0];
    k = dimensions[1];
    n = other.dimensions[1];

    float alpha = 1, beta = 0; // GEMM input parameters, C=α*AB+β*C

    float *dA, *dB, *dC;

    float* A = data.get();
    float* B = other.data.get();
    float* C = new float[m * n];

    hipMalloc((void **)&dA, sizeof(float) * m * k);
    hipMalloc((void **)&dB, sizeof(float) * k * n);
    hipMalloc((void **)&dC, sizeof(float) * m * n);

    hipMemcpy(dA, A, sizeof(float) * m * k, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(float) * k * n, hipMemcpyHostToDevice);
    hipMemcpy(dC, C, sizeof(float) * m * n, hipMemcpyHostToDevice);

    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, dB, HIP_R_32F,
                n, dA, HIP_R_32F, k, &beta, dC, HIP_R_32F, n, HIPBLAS_COMPUTE_32F,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    hipDeviceSynchronize();
    hipMemcpy(C, dC, sizeof(float) * m * n, hipMemcpyDeviceToHost);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipblasDestroy(handle);

    Tensor result = Tensor::empty({m, n});
    result.data = shared_ptr<float>(C, default_delete<float[]>());

    return result;
}
